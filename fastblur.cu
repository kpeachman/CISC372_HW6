#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <time.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"


__global__ void computeColumn(uint8_t* src,float* dest,int col,int pWidth,int height,int radius,int bpp){
    int i;
    //initialize the first element of each column
    dest[col]=src[col];
    //start tue sum up to radius*2 by only adding
    for (i=1;i<=radius*2;i++)
        dest[i*pWidth+col]=src[i*pWidth+col]+dest[(i-1)*pWidth+col];
    for (i=radius*2+1;i<height;i++)
        dest[i*pWidth+col]=src[i*pWidth+col]+dest[(i-1)*pWidth+col]-src[(i-2*radius-1)*pWidth+col];
    //now shift everything up by radius spaces and blank out the last radius items to account for sums at the end of the kernel, instead of the middle
    for (i=radius;i<height;i++){
        dest[(i-radius)*pWidth+col]=dest[i*pWidth+col]/(radius*2+1);
    }
    //now the first and last radius values make no sense, so blank them out
    for (i=0;i<radius;i++){
        dest[i*pWidth+col]=0;
        dest[(height-1)*pWidth-i*pWidth+col]=0;
    }

}


int main(int argc,char** argv){
    long t1,t2;
    int radius=0;
    int i;
    int width,height,bpp,pWidth;
    char* filename;
    uint8_t *img;
    float* dest,*mid;

    if (argc!=3)
        return Usage(argv[0]);
    filename=argv[1];
    sscanf(argv[2],"%d",&radius);
   
    img=stbi_load(filename,&width,&height,&bpp,0);

    pWidth=width*bpp;  //actual width in bytes of an image row

    hipMalloc(mid, sizeof(float)*pWidth*height);   
    hipMalloc(dest, sizeof(float)*pWidth*height);   

}